#include <vector>

#include "cuda/01_morton.cuh"
#include "cuda/02_sort.cuh"
#include "cuda/03_unique.cuh"
#include "cuda/04_radix_tree.cuh"
#include "cuda/05_edge_count.cuh"
#include "cuda/06_prefix_sum.cuh"
#include "cuda/07_octree.cuh"
#include "cuda/helper.cuh"
#include "shared/structures.h"

namespace gpu {

std::vector<hipStream_t> streams;  // need to initialize

void initialize_dispatcher(const int n_streams) {
  streams.resize(n_streams);
  for (int i = 0; i < n_streams; ++i) {
    CHECK_CUDA_CALL(hipStreamCreate(&streams[i]));
  }
}

void release_dispatcher() {
  for (auto stream : streams) {
    CHECK_CUDA_CALL(hipStreamDestroy(stream));
  }
}

void sync_device() { SYNC_DEVICE(); }

void sync_stream(const int stream_id) {
  CHECK_CUDA_CALL(hipStreamSynchronize(streams[stream_id]));
}

void dispatch_ComputeMorton(const int grid_size,
                            const int stream_id,
                            pipe& pipe) {
  constexpr auto block_size = 768;

  k_ComputeMortonCode<<<grid_size, block_size, 0, streams[stream_id]>>>(
      pipe.u_points, pipe.u_morton, pipe.n_input(), pipe.min_coord, pipe.range);
}

void dispatch_RadixSort(const int grid_size, const int stream_id, pipe& pipe);

void dispatch_RemoveDuplicates(const int grid_size,
                               const int stream_id,
                               pipe& pipe);

void dispatch_BuildRadixTree(const int grid_size,
                             const int stream_id,
                             pipe& pipe);

void dispatch_EdgeCount(const int grid_size, const int stream_id, pipe& pipe);

void dispatch_EdgeOffset(const int grid_size, const int stream_id, pipe& pipe);

void dispatch_BuildOctree(const int grid_size, const int stream_id, pipe& pipe);

}  // namespace gpu
